#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_ptr.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

struct is_terminated {
    __host__ __device__ bool operator()(const PathSegment& path) {
        return path.remainingBounces <= 0;
    }
};


__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
static EnvironmentMap dev_environmentMap;
static glm::vec3* dev_envmap_data = NULL;
static LightInfo* dev_lights = NULL;
static int num_lights = 0;

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

// Get the area of a geometry
__host__ __device__ float getGeomArea(const Geom& geom) {
    if (geom.type == SPHERE) {
        float radius = geom.scale.x * 0.5f;
        return 4.0f * PI * radius * radius;
    }
    else if (geom.type == CUBE) {
        float sx = geom.scale.x;
        float sy = geom.scale.y;
        float sz = geom.scale.z;
        return 2.0f * (sx * sy + sy * sz + sz * sx);
    }
    return 1.0f; // Default
}

void initializeLights(Scene* scene) {
    // Count emissive objects
    std::vector<LightInfo> lightInfos;
    float totalArea = 0.0f;

    for (int i = 0; i < scene->geoms.size(); i++) {
        int matId = scene->geoms[i].materialid;
        if (scene->materials[matId].emittance > 0.0f) {
            LightInfo info;
            info.geomIdx = i;
            info.area = getGeomArea(scene->geoms[i]);
            totalArea += info.area;
            lightInfos.push_back(info);
        }
    }

    // Normalize PDFs
    for (auto& light : lightInfos) {
        light.pdf = light.area / totalArea;
    }

    num_lights = lightInfos.size();
    if (num_lights > 0) {
        hipMalloc(&dev_lights, num_lights * sizeof(LightInfo));
        hipMemcpy(dev_lights, lightInfos.data(), num_lights * sizeof(LightInfo),
            hipMemcpyHostToDevice);
    }

    printf("Initialized %d light sources for MIS\n", num_lights);
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

	// Initialize lights for MIS
	initializeLights(scene);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need
    // Initialize environment map
    dev_environmentMap.enabled = scene->environmentMap.enabled;
    dev_environmentMap.width = scene->environmentMap.width;
    dev_environmentMap.height = scene->environmentMap.height;
    dev_environmentMap.intensity = scene->environmentMap.intensity;

    if (scene->environmentMap.enabled && scene->environmentMap.data.size() > 0)
    {
        size_t envMapSize = scene->environmentMap.width * scene->environmentMap.height * sizeof(glm::vec3);
        hipMalloc(&dev_envmap_data, envMapSize);
        hipMemcpy(dev_envmap_data, scene->environmentMap.data.data(), envMapSize, hipMemcpyHostToDevice);
        dev_environmentMap.data = dev_envmap_data;

        //printf("Environment map uploaded to GPU: %dx%d pixels, %.2f MB\n",
        //    scene->environmentMap.width, scene->environmentMap.height,
        //    envMapSize / (1024.0f * 1024.0f));
    }
    else
    {
        dev_environmentMap.data = nullptr;
    }

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
    // Free environment map data
    if (dev_envmap_data != NULL)
    {
        hipFree(dev_envmap_data);
        dev_envmap_data = NULL;
    }

    if (dev_lights != NULL) {
        hipFree(dev_lights);
        dev_lights = NULL;
    }

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        // TODO: implement antialiasing by jittering the ray
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
        );

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
        }
    }
}

// ===== DEVICE FUNCTIONS FOR ENVIRONMENT MAP =====

__device__ glm::vec3 sampleEnvironmentMap(const glm::vec3& direction, const EnvironmentMap& envMap)
{
    if (!envMap.enabled || envMap.data == nullptr) {
        return glm::vec3(0.0f);
    }

    // Convert direction to spherical coordinates
    // theta: angle from +Y axis (0 to PI)
    // phi: angle around Y axis from +X (0 to 2*PI)
    float theta = acosf(fmaxf(-1.0f, fminf(1.0f, direction.y)));
    float phi = atan2f(direction.z, direction.x);

    // Convert to UV coordinates [0, 1]
    float u = (phi + PI) / (2.0f * PI);
    float v = theta / PI;

    // Clamp UV to valid range
    u = fmaxf(0.0f, fminf(1.0f, u));
    v = fmaxf(0.0f, fminf(1.0f, v));

    // Convert to pixel coordinates
    float fx = u * (envMap.width - 1);
    float fy = v * (envMap.height - 1);

    // Bilinear interpolation for smoother sampling
    int x0 = (int)floorf(fx);
    int y0 = (int)floorf(fy);
    int x1 = min(x0 + 1, envMap.width - 1);
    int y1 = min(y0 + 1, envMap.height - 1);

    float wx = fx - x0;
    float wy = fy - y0;

    // Sample four neighboring pixels
    glm::vec3 p00 = envMap.data[y0 * envMap.width + x0];
    glm::vec3 p10 = envMap.data[y0 * envMap.width + x1];
    glm::vec3 p01 = envMap.data[y1 * envMap.width + x0];
    glm::vec3 p11 = envMap.data[y1 * envMap.width + x1];

    // Bilinear interpolation
    glm::vec3 p0 = p00 * (1.0f - wx) + p10 * wx;
    glm::vec3 p1 = p01 * (1.0f - wx) + p11 * wx;
    glm::vec3 result = p0 * (1.0f - wy) + p1 * wy;

    return result;
}

// ===== HELPER FUNCTIONS FOR MIS =====

// Power heuristic for MIS (balance heuristic with beta=2)
__device__ float powerHeuristic(float pdfA, float pdfB) {
    // FIREFLY FIX: Ensure non-zero PDFs
    pdfA = max(pdfA, 1e-8f);
    pdfB = max(pdfB, 1e-8f);

    float pdfA2 = pdfA * pdfA;
    float pdfB2 = pdfB * pdfB;
    return pdfA2 / (pdfA2 + pdfB2);
}

// Sample a point on a sphere
__device__ glm::vec3 sampleSphere(const Geom& geom, thrust::default_random_engine& rng) {
    thrust::uniform_real_distribution<float> u01(0, 1);

    float u = u01(rng);
    float v = u01(rng);

    float theta = 2.0f * PI * u;
    float phi = acos(1.0f - 2.0f * v);

    float radius = geom.scale.x * 0.5f; // Assuming uniform scale for sphere

    glm::vec3 local(
        radius * sin(phi) * cos(theta),
        radius * sin(phi) * sin(theta),
        radius * cos(phi)
    );

    return glm::vec3(geom.transform * glm::vec4(local, 1.0f));
}

// Sample a point on a box
__device__ glm::vec3 sampleBox(const Geom& geom, thrust::default_random_engine& rng) {
    thrust::uniform_real_distribution<float> u01(0, 1);

    // Choose which face to sample
    float faceChoice = u01(rng) * 6.0f;
    int face = (int)faceChoice;

    float u = u01(rng) - 0.5f;
    float v = u01(rng) - 0.5f;

    glm::vec3 local;
    switch (face) {
    case 0: local = glm::vec3(0.5f, u, v); break;   // +X
    case 1: local = glm::vec3(-0.5f, u, v); break;  // -X
    case 2: local = glm::vec3(u, 0.5f, v); break;   // +Y
    case 3: local = glm::vec3(u, -0.5f, v); break;  // -Y
    case 4: local = glm::vec3(u, v, 0.5f); break;   // +Z
    default: local = glm::vec3(u, v, -0.5f); break; // -Z
    }

    return glm::vec3(geom.transform * glm::vec4(local, 1.0f));
}

// Sample a point on any light source
__device__ glm::vec3 sampleLight(const Geom& geom, thrust::default_random_engine& rng) {
    if (geom.type == SPHERE) {
        return sampleSphere(geom, rng);
    }
    else if (geom.type == CUBE) {
        return sampleBox(geom, rng);
    }
    return geom.translation; // Fallback to center
}

// ===== SHADING HELPER FUNCTIONS =====
__host__ __device__ void shadeDiffuse(
    PathSegment& pathSegment,
    const ShadeableIntersection& intersection,
    glm::vec3 materialColor,
    thrust::default_random_engine rng
    ) {
    

    // Generate new ray direction using cosine-weighted sampling
    glm::vec3 wiW = calculateRandomDirectionInHemisphere(
        intersection.surfaceNormal, rng);

    // For cosine-weighted sampling with Lambertian BRDF:
    // The math simplifies to just multiplying by the material color
    pathSegment.color *= materialColor;

    // Set up the new ray
    glm::vec3 intersectionPoint = pathSegment.ray.origin +
        pathSegment.ray.direction * intersection.t;
    pathSegment.ray.origin = intersectionPoint +
        intersection.surfaceNormal * 0.001f;
    pathSegment.ray.direction = wiW;
}

// ===== MIS DIFFUSE SHADING WITH DIRECT LIGHTING =====
__device__ glm::vec3 clamp(const glm::vec3& v, const glm::vec3& min, const glm::vec3& max) {
    return glm::vec3(
        fminf(fmaxf(v.x, min.x), max.x),
        fminf(fmaxf(v.y, min.y), max.y),
        fminf(fmaxf(v.z, min.z), max.z)
    );
}

__device__ float clamp(float v, float min, float max) {
    return fminf(fmaxf(v, min), max);
}

__device__ void shadeDiffuseMIS(
    PathSegment& pathSegment,
    const ShadeableIntersection& intersection,
    glm::vec3 materialColor,
    Geom* geoms,
    int num_geoms,
    Material* materials,
    LightInfo* lights,
    int num_lights,
    thrust::default_random_engine& rng
) {
    if (num_lights == 0) {
        shadeDiffuse(pathSegment, intersection, materialColor, rng);
        return;
    }

    thrust::uniform_real_distribution<float> u01(0, 1);

    glm::vec3 intersectionPoint = pathSegment.ray.origin +
        pathSegment.ray.direction * intersection.t;
    glm::vec3 normal = intersection.surfaceNormal;

    glm::vec3 totalContribution(0.0f);

    // === DIRECT LIGHTING (Light Sampling) ===

    // Randomly select a light
    int lightIdx = (int)(u01(rng) * num_lights);
    lightIdx = min(lightIdx, num_lights - 1);
    LightInfo& lightInfo = lights[lightIdx];
    Geom& lightGeom = geoms[lightInfo.geomIdx];
    Material& lightMat = materials[lightGeom.materialid];

    // Sample a point on the light
    glm::vec3 lightPoint = sampleLight(lightGeom, rng);
    glm::vec3 toLight = lightPoint - intersectionPoint;
    float distToLight = glm::length(toLight);

    // FIREFLY FIX #1: Skip if too close to avoid numerical issues
    if (distToLight < 0.01f) {
        shadeDiffuse(pathSegment, intersection, materialColor, rng);
        return;
    }

    toLight = glm::normalize(toLight);

    // Check visibility (shadow ray)
    Ray shadowRay;
    shadowRay.origin = intersectionPoint + normal * 0.001f;
    shadowRay.direction = toLight;

    bool visible = true;
    glm::vec3 tmp_intersect, tmp_normal;
    bool tmp_outside;

    // Check intersection with all geometries
    for (int i = 0; i < num_geoms; i++) {
        if (i == lightInfo.geomIdx) continue; // Skip the light itself

        float t = -1.0f;
        if (geoms[i].type == CUBE) {
            t = boxIntersectionTest(geoms[i], shadowRay, tmp_intersect, tmp_normal, tmp_outside);
        }
        else if (geoms[i].type == SPHERE) {
            t = sphereIntersectionTest(geoms[i], shadowRay, tmp_intersect, tmp_normal, tmp_outside);
        }

        if (t > 0.0f && t < distToLight - 0.001f) {
            visible = false;
            break;
        }
    }

    if (visible) {
        // Compute light normal at sampled point (approximate)
        glm::vec3 lightNormal = glm::normalize(lightPoint - lightGeom.translation);

        // Geometric term
        float cosThetaLight = abs(glm::dot(-toLight, lightNormal));
        float cosThetaSurface = max(0.0f, glm::dot(toLight, normal));

        // FIREFLY FIX #2: Clamp distance squared to avoid extreme values
        float distSquaredClamped = max(distToLight * distToLight, 0.1f);
        float geometricTerm = cosThetaSurface * cosThetaLight / distSquaredClamped;

        // FIREFLY FIX #3: Ensure minimum area to avoid huge PDFs
        float safeArea = max(lightInfo.area, 0.01f);

        // PDFs for MIS
        float pdfLight = (1.0f / safeArea) * (1.0f / num_lights);
        float pdfBRDF = cosThetaSurface / PI;

        // FIREFLY FIX #4: Clamp PDFs to reasonable range
        pdfLight = clamp(pdfLight, 0.001f, 1000.0f);
        pdfBRDF = max(pdfBRDF, 0.001f);

        // FIREFLY FIX #5: Use balance heuristic for small lights (more stable)
        float misWeight;
        if (lightInfo.area < 0.1f) {
            // Balance heuristic for small lights
            misWeight = pdfLight / (pdfLight + pdfBRDF);
        }
        else {
            // Power heuristic for normal lights
            float pdfLight2 = pdfLight * pdfLight;
            float pdfBRDF2 = pdfBRDF * pdfBRDF;
            misWeight = pdfLight2 / (pdfLight2 + pdfBRDF2);
        }

        // Direct lighting contribution
        glm::vec3 lightEmission = lightMat.color * lightMat.emittance;
        glm::vec3 brdf = materialColor / PI;

        glm::vec3 contribution = misWeight * lightEmission * brdf * geometricTerm / pdfLight;

        // FIREFLY FIX #6: Clamp final contribution to prevent fireflies
        const float MAX_CONTRIBUTION = 10.0f;  // Tune this based on scenes
        contribution = clamp(contribution, glm::vec3(0.0f), glm::vec3(MAX_CONTRIBUTION));

        totalContribution += contribution;
    }

    // === INDIRECT LIGHTING (BRDF Sampling) ===

    // Sample the BRDF (cosine-weighted hemisphere)
    glm::vec3 wiW = calculateRandomDirectionInHemisphere(normal, rng);

    // Update path color with direct lighting contribution
    pathSegment.color *= materialColor + totalContribution;

    // Set up the new ray for indirect lighting
    pathSegment.ray.origin = intersectionPoint + normal * 0.001f;
    pathSegment.ray.direction = wiW;
}

__host__ __device__ void shadeSpecular(
    PathSegment& pathSegment,
    const ShadeableIntersection& intersection,
    glm::vec3 materialColor
    ) 
{
    // Perfectly specular reflection direction
    glm::vec3 normal = intersection.surfaceNormal;
    glm::vec3 incident = pathSegment.ray.direction;
    glm::vec3 reflected = glm::normalize(incident - 2.0f * glm::dot(incident, normal) * normal);
    // For perfect specular reflection with ideal mirror BRDF:
    // The math simplifies to just multiplying by the material color
    pathSegment.color *= materialColor;
    // Set up the new ray
    glm::vec3 intersectionPoint = pathSegment.ray.origin +
        pathSegment.ray.direction * intersection.t;
    pathSegment.ray.origin = intersectionPoint +
        intersection.surfaceNormal * 0.001f;
    pathSegment.ray.direction = reflected;
}

__host__ __device__ float shlickFresnel(float cosTheta, float ior) {
    float r0 = (1.0f - ior) / (1.0f + ior);
    r0 = r0 * r0;
    return r0 + (1.0f - r0) * pow((1.0f - cosTheta), 5.0f);
}

__host__ __device__ void shadeRefractive(
    PathSegment& pathSegment,
    const ShadeableIntersection& intersection,
    glm::vec3 materialColor,
    float ior,
	thrust::default_random_engine& rng
)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

	glm::vec3 normal = intersection.surfaceNormal;
	glm::vec3 incident = glm::normalize(pathSegment.ray.direction);

	float cosTheta = glm::dot(incident, normal);
	bool entering = cosTheta < 0.0f;

    float etaI, etaT;
	if (entering) {
        etaI = 1.0f; // air
        etaT = ior;
        cosTheta = -cosTheta; 
    }
    else {
        etaI = ior;
        etaT = 1.0f; // air
        normal = -normal;
    }

	float eta = etaI / etaT;

	float sin2ThetaT = eta * eta * (1.0f - cosTheta * cosTheta);

	glm::vec3 newDirection;

    if (sin2ThetaT > 1.0f) {
        // Total internal reflection
		newDirection = glm::normalize(incident - 2.0f * glm::dot(incident, normal) * normal);
    }
    else {
        float cosThetaT = sqrtf(1.0f - sin2ThetaT);
		float fresnelReflectance = shlickFresnel(entering ? cosTheta : cosThetaT, eta);

        if (u01(rng) < fresnelReflectance) {
			// Reflect
			newDirection = glm::reflect(incident, normal);
        }
        else {
			newDirection = glm::normalize(eta * incident + (eta * cosTheta - cosThetaT) * normal);
        }
	}

	pathSegment.color *= materialColor;

	// Set up the new ray
    glm::vec3 intersectionPoint = pathSegment.ray.origin +
		pathSegment.ray.direction * intersection.t;

    pathSegment.ray.origin = intersectionPoint + newDirection * 0.001f;
	pathSegment.ray.direction = newDirection;
}

// ===== MAIN SHADING KERNEL =====
__global__ void shadeMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
	EnvironmentMap envMap,
    bool firstIter)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    // Skip already terminated paths
    if (pathSegments[idx].remainingBounces <= 0) {
        return;
    }

    ShadeableIntersection intersection = shadeableIntersections[idx];

    if (intersection.t > 0.0f) // Ray hit something
    {
        

        Material material = materials[intersection.materialId];
        glm::vec3 materialColor = material.color;

        // Handle light sources
        if (material.emittance > 0.0f) {
            // Ray hit a light - accumulate emission and terminate
            pathSegments[idx].color *= (materialColor * material.emittance);
            pathSegments[idx].remainingBounces = 0;
            return;
        }

        // Handle diffuse materials
        // Decrement bounces first
        pathSegments[idx].remainingBounces--;

        // Check if we should continue
        if (pathSegments[idx].remainingBounces <= 0) {
            // Maximum depth reached without hitting light
            // The path contributes nothing (black)
            pathSegments[idx].color = glm::vec3(0.0f);
            return;
        }

        // Set up RNG with proper seed
        thrust::default_random_engine rng = makeSeededRandomEngine(
            iter, idx, pathSegments[idx].remainingBounces);
        thrust::uniform_real_distribution<float> u01(0, 1);

		MaterialType mType = material.type;

        switch (mType) {
        case DIFFUSE:
			pathSegments[idx].prevIsSpecular = false;
            // TODO: implement MIS
            shadeDiffuse(pathSegments[idx], intersection, materialColor, rng);
            break;

		case SPECULAR:
            pathSegments[idx].prevIsSpecular = true;
			shadeSpecular(pathSegments[idx], intersection, materialColor);
			break;

		case REFRACTIVE:
            pathSegments[idx].prevIsSpecular = true;
			shadeRefractive(pathSegments[idx], intersection, materialColor, material.indexOfRefraction, rng);
			break;

        default:
            pathSegments[idx].prevIsSpecular = false;
            shadeDiffuse(pathSegments[idx], intersection, materialColor, rng);
            break;
        }

    }
    else {
        

        if (envMap.enabled) {
            glm::vec3 envColor = sampleEnvironmentMap(pathSegments[idx].ray.direction, envMap);
            if (firstIter) {
                // Direct visibility of environment
                pathSegments[idx].color *= envColor;
            }
            else if (pathSegments[idx].prevIsSpecular) {
                // Environment visible through reflection/refraction
                pathSegments[idx].color *= envColor;
            }
            else {
                // Diffuse bounce missed - could use ambient or black
                // Using environment as ambient light
                pathSegments[idx].color *= envColor * 0.5f; // Reduced contribution
            }
        }
        else {
            // No environment map - use black
            pathSegments[idx].color *= glm::vec3(0.0f);
        }

        //pathSegments[idx].color = glm::vec3(0.0f);
        pathSegments[idx].remainingBounces = 0;
    }
}

// ===== MODIFIED SHADING KERNEL =====

__global__ void shadeMaterialMIS(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
    Geom* geoms,
    int num_geoms,
    LightInfo* lights,
    int num_lights,
    EnvironmentMap envMap,
    bool firstIter)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths) return;

    if (pathSegments[idx].remainingBounces <= 0) {
        return;
    }

    ShadeableIntersection intersection = shadeableIntersections[idx];

    if (intersection.t > 0.0f) {
        Material material = materials[intersection.materialId];
        glm::vec3 materialColor = material.color;

        // Handle light sources
        if (material.emittance > 0.0f) {
            pathSegments[idx].color *= (materialColor * material.emittance);
            pathSegments[idx].remainingBounces = 0;
            return;
        }

        pathSegments[idx].remainingBounces--;

        if (pathSegments[idx].remainingBounces <= 0) {
            pathSegments[idx].color = glm::vec3(0.0f);
            return;
        }

        thrust::default_random_engine rng = makeSeededRandomEngine(
            iter, idx, pathSegments[idx].remainingBounces);

        MaterialType mType = material.type;

        switch (mType) {
        case DIFFUSE:
            pathSegments[idx].prevIsSpecular = false;
            // Use MIS for diffuse materials
            shadeDiffuseMIS(pathSegments[idx], intersection, materialColor,
                geoms, num_geoms, materials, lights, num_lights, rng);
            break;

        case SPECULAR:
            pathSegments[idx].prevIsSpecular = true;
            shadeSpecular(pathSegments[idx], intersection, materialColor);
            break;

        case REFRACTIVE:
            pathSegments[idx].prevIsSpecular = true;
            shadeRefractive(pathSegments[idx], intersection, materialColor,
                material.indexOfRefraction, rng);
            break;

        default:
            pathSegments[idx].prevIsSpecular = false;
            shadeDiffuseMIS(pathSegments[idx], intersection, materialColor,
                geoms, num_geoms, materials, lights, num_lights, rng);
            break;
        }
    }
    else {
        // Handle environment map or background
        if (envMap.enabled) {
            glm::vec3 envColor = sampleEnvironmentMap(pathSegments[idx].ray.direction, envMap);
            if (firstIter) {
                pathSegments[idx].color *= envColor;
            }
            else if (pathSegments[idx].prevIsSpecular) {
                pathSegments[idx].color *= envColor;
            }
            else {
                pathSegments[idx].color *= envColor * 0.5f;
            }
        }
        else {
            pathSegments[idx].color *= glm::vec3(0.0f);
        }
        pathSegments[idx].remainingBounces = 0;
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

__global__ void gatherTerminatedPaths(int nPaths, glm::vec3* image, PathSegment* paths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment path = paths[index];
        // Only add to image if this path is terminated
        if (path.remainingBounces == 0) {
            image[path.pixelIndex] += path.color;
        }
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool firstIter = true;
    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.

   //     shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
   //         iter,
   //         num_paths,
   //         dev_intersections,
   //         dev_paths,
   //         dev_materials,
			//dev_environmentMap,
   //         firstIter
   //     );
        shadeMaterialMIS << <numblocksPathSegmentTracing, blockSize1d >> > (
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_lights,
            num_lights,
            dev_environmentMap,
            firstIter
            );
		hipDeviceSynchronize();

        if (firstIter) {
            firstIter = false;
		}

        dim3 numBlocksGather = (num_paths + blockSize1d - 1) / blockSize1d;
        gatherTerminatedPaths << <numBlocksGather, blockSize1d >> > (
            num_paths,
            dev_image,
            dev_paths
            );
        checkCUDAError("gather terminated paths");

        // --- Stream Compaction Stage ---
        PathSegment* new_end = thrust::remove_if(thrust::device,
            dev_paths,
            dev_paths + num_paths,
            is_terminated());

        int paths_before = num_paths;
        num_paths = new_end - dev_paths;

        // Check termination conditions
        if (num_paths == 0 || depth >= traceDepth) {
            iterationComplete = true;
        }

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
